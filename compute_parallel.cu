#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <hip/hip_runtime.h>

extern vector3 *all_values;
extern double *d_hPos, *d_hVel, *d_mass;
//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL

__global__ void compute_kernel(vector3 *values, double *hPos, double *hVel, double *mass){
	int i,j,k;

	//	vector3** accels=(vector3**)malloc(sizeof(vector3*)*NUMENTITIES);
	// vector3* values=(vector3*)malloc(sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	// for (i=0;i<NUMENTITIES;i++) // what is this doing
	//	accels[i]=&values[i*NUMENTITIES];
	
	
	//first compute the pairwise accelerations.  Effect is on the first argument.

	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;

	// must stay in bounds
	if (i >= NUMENTITIES || j >= NUMENTITIES){
		return;
	}
	// planet goes not have affect on itself
	if (i==j) {
		FILL_VECTOR(values[i*NUMENTITIES + j],0,0,0);
	}
	else{
		vector3 distance;
		for (k=0;k<3;k++) distance[k]=hPos[i*3 + k]-hPos[j*3 + k];
		double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
		double magnitude=sqrt(magnitude_sq);
		double accelmag=-1*GRAV_CONSTANT*mass[j]/magnitude_sq;
		FILL_VECTOR(values[i*NUMENTITIES + j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
	}
}

__global__ void add_kernel(vector3 *values, double *hPos, double *hVel, double *mass){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j, k;

	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	
	if (i > NUMENTITIES) return;

	vector3 accel_sum={0,0,0};
	for (j=0;j<NUMENTITIES;j++){
		for (k=0;k<3;k++)
			accel_sum[k]+=values[i*NUMENTITIES + j][k];
	}

	//compute the new velocity based on the acceleration and time interval
	//compute the new position based on the velocity and time interval
	for (k=0;k<3;k++){
		hVel[i*3 + k]+=accel_sum[k]*INTERVAL;
		hPos[i*3 + k]+=hVel[i*3 + k]*INTERVAL;
	}
		
}


void compute(){
	
	// creating variables (host)


	// kernel call for compute
	dim3 dimBlock(16, 16);
	dim3 dimGrid((NUMENTITIES + 15)/16, (NUMENTITIES + 15)/16);
	compute_kernel<<<dimGrid, dimBlock>>>(all_values, d_hPos, d_hVel, d_mass);
	hipDeviceSynchronize();

	// kernel call for add
	add_kernel<<<NUMENTITIES, 1>>>(all_values, d_hPos, d_hVel, d_mass);
	hipDeviceSynchronize();
	
	// what is the reduciton portion?? is it just the addition

	
	//free(accels);

}
